#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

__global__ void findMax(int N, int* thelistArray)
{
	int index = threadIdx.x;
	int numthreads = blockDim.x;
	int max = thelistArray[0];
	for (int i = index; i < N; i += numthreads)
	{
		if (max < thelistArray[i])
		{
			max = thelistArray[i];
		}
	}

	thelistArray[0] = max;
}

hipEvent_t beginEvent1;
hipEvent_t endEvent1;
hipEvent_t beginEvent2;
hipEvent_t endEvent2;

int main(void)
{
	int input = 0;
	std::cout << "Input: ";
	std::cin >> input;
	thrust::host_vector<int> thelist;
	for (int i = 0; i < input; i++)
	{
		thelist.push_back(rand() % 1000 + 1);
	}

	//CPU-METHOD
	hipEventCreate(&beginEvent1);
	hipEventCreate(&endEvent1);
	hipEventRecord(beginEvent1, 0);

	int largest = thelist[0];
	for (int j = 1; j < input; j++)
	{
		if (thelist[j] > largest)
		{
			largest = thelist[j];
		}
	}
	hipEventRecord(endEvent1, 0);
	hipEventSynchronize(endEvent1);
	float time;
	hipEventElapsedTime(&time, beginEvent1, endEvent1);
	std::cout << "Value: " << largest << "\n";
	std::cout << "Time: " << time << " milliseconds" << "\n";
	hipEventDestroy(beginEvent1);
	hipEventDestroy(endEvent1);

	//GPU-METHOD
	hipEventCreate(&beginEvent2);
	hipEventCreate(&endEvent2);
	hipEventRecord(beginEvent2, 0);
	int* thelistArray = thrust::raw_pointer_cast(&thelist[0]);
	findMax <<<1, 256 >>> (input, thelistArray);
	hipEventRecord(endEvent2, 0);
	hipEventSynchronize(endEvent2);
	float time2;
	hipEventElapsedTime(&time2, beginEvent2, endEvent2);
	std::cout << "Value: " << thelistArray[0] << "\n";
	std::cout << "Time: " << time2 << " milliseconds" << "\n";
	hipEventDestroy(beginEvent2);
	hipEventDestroy(endEvent2);
}
